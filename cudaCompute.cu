#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

// taken from Dr.Dobbs
// http://www.ddj.com/cpp/207200659

// the next line was changed from
// void cudaCompute(void)
// to 
// extern "C" void cudaCompute(void)

extern "C" void cudaCompute(void)
{
   float *a_h, *b_h;     // pointers to host memory
   float *a_d, *b_d;     // pointers to device memory
   int N = 5;
   int i;
   
   // allocate arrays on host
   a_h = (float *)malloc(sizeof(float)*N);
   b_h = (float *)malloc(sizeof(float)*N);
   
   // allocate arrays on device
   hipMalloc((void **) &a_d, sizeof(float)*N);
   hipMalloc((void **) &b_d, sizeof(float)*N);
   
   // initialize host data
   printf("initialize host data\n");
   for (i=0; i<N; i++) {
      a_h[i] = 10.f+i; 	// a = 10 to 14
      b_h[i] = 0.f;		// b = 0
      printf(" a_h[%d] = %f\t b_h[%d] = %f\n", i, a_h[i], i, b_h[i]);
   }
   
   // send data from host to device: a_h to a_d
   // target, source, size, direction
   hipMemcpy(a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
   
   // copy data within device: a_d to b_d
   hipMemcpy(b_d, a_d, sizeof(float)*N, hipMemcpyDeviceToDevice);
   
   // retrieve data from device: b_d to b_h
   hipMemcpy(b_h, b_d, sizeof(float)*N, hipMemcpyDeviceToHost);
   
   // check result
   printf("assert received data\n");
   for (i=0; i<N; i++) {
      assert(a_h[i] == b_h[i]);
      // if correct a_h = 10 to 14 
      // if correct b_h = 10 to 14
      printf(" a_h[%d] = %f\t b_h[%d] = %f\n", i, a_h[i], i, b_h[i]); 
   }
   
   // cleanup
   free(a_h); free(b_h); 
   hipFree(a_d); hipFree(b_d);
}